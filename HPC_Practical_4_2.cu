#include "hip/hip_runtime.h"
//Write a cuda program for
//2. Matrics Multiplication using CUDA C

#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024

__global__ void multiplyMatrices(int *A, int *B, int *C, int N) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < N && col < N) {
        int sum = 0;
        for (int k = 0; k < N; k++) sum += A[row * N + k] * B[k * N + col];
        C[row * N + col] = sum;
    }
}

int main() {
    int *A, *B, *C, *d_A, *d_B, *d_C;
    A = new int[N*N]; B = new int[N*N]; C = new int[N*N];

    for (int i = 0; i < N * N; i++) { A[i] = 1; B[i] = 1; }

    hipMalloc(&d_A, N * N * sizeof(int));
    hipMalloc(&d_B, N * N * sizeof(int));
    hipMalloc(&d_C, N * N * sizeof(int));

    hipMemcpy(d_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);
    multiplyMatrices<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, N * N * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "C[0] = " << C[0] << ", C[1] = " << C[1] << std::endl;  // Example output
    delete[] A; delete[] B; delete[] C;
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    return 0;
}
