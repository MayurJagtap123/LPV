//Write a cuda program for
//1.Addition of two large vectors

#include <iostream>
#include <hip/hip_runtime.h>

__global__ void addVectors(int *A, int *B, int *C, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) C[idx] = A[idx] + B[idx];
}

int main() {
    int N = 1000000;
    int *A, *B, *C, *d_A, *d_B, *d_C;
    A = new int[N]; B = new int[N]; C = new int[N];

    for (int i = 0; i < N; i++) { A[i] = i; B[i] = i; }

    hipMalloc(&d_A, N * sizeof(int));
    hipMalloc(&d_B, N * sizeof(int));
    hipMalloc(&d_C, N * sizeof(int));

    hipMemcpy(d_A, A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(int), hipMemcpyHostToDevice);

    addVectors<<<(N + 255) / 256, 256>>>(d_A, d_B, d_C, N);
    
    hipMemcpy(C, d_C, N * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "C[0] = " << C[0] << ", C[1] = " << C[1] << std::endl;  // Example output
    delete[] A; delete[] B; delete[] C;
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    return 0;
}
