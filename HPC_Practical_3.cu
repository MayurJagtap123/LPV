//Implement Min, Max, Sum and Average oprations using Parallel Reduction.


#include <iostream>
#include <hip/hip_runtime.h>

__global__ void reduceMinMaxSum(int *a, int *mn, int *mx, int *sm) {
    __shared__ int smin[256], smax[256], ssum[256];
    int t = threadIdx.x;
    smin[t] = smax[t] = ssum[t] = a[t];
    __syncthreads();
    for (int s = 128; s > 0; s >>= 1) {
        if (t < s) {
            smin[t] = min(smin[t], smin[t + s]);
            smax[t] = max(smax[t], smax[t + s]);
            ssum[t] += ssum[t + s];
        }
        __syncthreads();
    }
    if (t == 0) *mn = smin[0], *mx = smax[0], *sm = ssum[0];
}

int main() {
    int h[256], *d, *mn, *mx, *sm, x, y, z;
    for (int i = 0; i < 256; i++) h[i] = rand() % 100;

    hipMalloc(&d, 256 * 4); hipMalloc(&mn, 4); hipMalloc(&mx, 4); hipMalloc(&sm, 4);
    hipMemcpy(d, h, 256 * 4, hipMemcpyHostToDevice);
    reduceMinMaxSum<<<1, 256>>>(d, mn, mx, sm);
    hipMemcpy(&x, mn, 4, hipMemcpyDeviceToHost);
    hipMemcpy(&y, mx, 4, hipMemcpyDeviceToHost);
    hipMemcpy(&z, sm, 4, hipMemcpyDeviceToHost);

    std::cout << "Min: " << x << "\nMax: " << y << "\nSum: " << z << "\nAvg: " << z / 256.0f << "\n";
    hipFree(d); hipFree(mn); hipFree(mx); hipFree(sm);
    return 0;
}

//nvcc -o reduction reduction.cu
//./reduction